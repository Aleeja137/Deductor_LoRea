#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h> 
#include <hip/hip_runtime.h>

__global__ void unifier_matrices_cuda(int *mat0, int *mat1, int n0, int n1, int *unifiers){

    // Launch n0*n1 threads in 1D, unifiers must have space for all threads to store a unifier
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i, j, m, row_size, unifier_size, code, last_unifier;
    int *unifier;

    m = mat0[0];
    last_unifier = 0;
    row_size = 1+m;
    unifier_size = 1+(2*m)+2;

    unifier = (int*) malloc (unifier_size*sizeof(int));

    i = gid/n1;
    j = gid%n1;
    
    memset(unifier,0,unifier_size*sizeof(int));  
    code = unifier_rows(&mat0[row_size * i], &mat1[row_size * j], unifier);
    if (code != 0) continue; // Rows cannot be unified

    code = correct_unifier(&mat0[row_size * i], &mat1[row_size * j], unifier);
    if (code != 0) continue; // Rows cannot be unified
    
    unifier[1+(2*m)]   = i;
    unifier[1+(2*m)+1] = j;

    // Here another kernel or host function will need to compress unifiers list
    memcpy(&unifiers[gid*unifier_size],unifier,unifier_size*sizeof(int));
    last_unifier++;

    // Free the extra space
    // unifiers = realloc(unifiers,last_unifier*unifier_size*sizeof(int));
    // free(unifier);
}


__global__ void unify_matrices_cuda(int *unified){
    // Launch n0 threads in 1D, each thread will handle one unifier
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    int line_len = 1+m0;
    int *line_A  = (int*) malloc (line_len*sizeof(int));
    int *line_B  = (int*) malloc (line_len*sizeof(int));
    // int *unified = (int*) malloc (unif_count*line_len*sizeof(int));

    int i = gid;
    int ind_A, ind_B;

    if (i < unif_count)
    {
        ind_A = unifiers[i*unifier_size+unifier_size-2];
        ind_B = unifiers[i*unifier_size+unifier_size-1];
        memcpy(line_A,&mat0[ind_A*line_len],line_len*sizeof(int));
        memcpy(line_B,&mat1[ind_B*line_len],line_len*sizeof(int));
        apply_unifier(line_A,line_B,&unifiers[i*unifier_size]);
        memcpy(&unified[i*line_len],line_A,line_len*sizeof(int));
    }

}
int main() {
    
    int n_blk = 1;
    int threads_per_blk = 1;

    apply_unifier_cuda <<<n_blk,threads_per_blk>>>(); 
    unify_matrices_cuda<<<n_blk,threads_per_blk>>>(); 

    return 0;
}